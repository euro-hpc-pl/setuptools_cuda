
#include <hip/hip_runtime.h>
template <typename T>
__global__ void _saxpy(T a, T* x, T* y, int n)
{
  int stride = gridDim.x * blockDim.x;
  int id = blockIdx.x * blockDim.x + threadIdx.x;
  for(int i = id; i < n; i += stride)
  {
    y[i] = a * x[i] + y[i];
  }
}

template <typename T>
void saxpy_wrapper(T a, T* x, T* y, int n, int numThreads, int numBlocks)
{
  T *dx, *dy;

  hipMalloc(&dx, n * sizeof(T));
  hipMalloc(&dy, n * sizeof(T));

  hipMemcpy(dx, x, n * sizeof(T), hipMemcpyHostToDevice);
  hipMemcpy(dy, y, n * sizeof(T), hipMemcpyHostToDevice);

  _saxpy<<<numThreads, numBlocks>>>(a, dx, dy, n);

  hipMemcpy(y, dy, n * sizeof(T), hipMemcpyDeviceToHost);

  hipFree(dx);
  hipFree(dy);
}

template void saxpy_wrapper(float, float*, float*, int, int, int);
template void saxpy_wrapper(double, double*, double*, int, int, int);
